#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/unique_and_compact_impl.cu
 * @brief Unique and compact operator implementation on CUDA.
 */
#include <graphbolt/cuda_ops.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/logical.h>

#include <hipcub/hipcub.hpp>
#include <type_traits>

#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

template <typename scalar_t>
struct EqualityFunc {
  const scalar_t* sorted_order;
  const scalar_t* found_locations;
  const scalar_t* searched_items;
  __host__ __device__ auto operator()(int64_t i) {
    return sorted_order[found_locations[i]] == searched_items[i];
  }
};

#define DefineCubReductionFunction(cub_reduce_fn, name)           \
  template <typename scalar_iterator_t>                           \
  auto name(const scalar_iterator_t input, int64_t size) {        \
    using scalar_t = std::remove_reference_t<decltype(input[0])>; \
    cuda::CopyScalar<scalar_t> result;                            \
    CUB_CALL(cub_reduce_fn, input, result.get(), size);           \
    return result;                                                \
  }

DefineCubReductionFunction(DeviceReduce::Max, Max);
DefineCubReductionFunction(DeviceReduce::Min, Min);

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> UniqueAndCompact(
    const torch::Tensor src_ids, const torch::Tensor dst_ids,
    const torch::Tensor unique_dst_ids, int num_bits) {
  TORCH_CHECK(
      src_ids.scalar_type() == dst_ids.scalar_type() &&
          dst_ids.scalar_type() == unique_dst_ids.scalar_type(),
      "Dtypes of tensors passed to UniqueAndCompact need to be identical.");
  auto allocator = cuda::GetAllocator();
  auto stream = cuda::GetCurrentStream();
  return AT_DISPATCH_INTEGRAL_TYPES(
      src_ids.scalar_type(), "unique_and_compact", ([&] {
        auto src_ids_ptr = src_ids.data_ptr<scalar_t>();
        auto dst_ids_ptr = dst_ids.data_ptr<scalar_t>();
        auto unique_dst_ids_ptr = unique_dst_ids.data_ptr<scalar_t>();

        // If num_bits is not given, compute maximum vertex ids to compute
        // num_bits later to speedup the expensive sort operations.
        cuda::CopyScalar<scalar_t> max_id_src;
        cuda::CopyScalar<scalar_t> max_id_dst;
        if (num_bits == 0) {
          max_id_src = Max(src_ids_ptr, src_ids.size(0));
          max_id_dst = Max(unique_dst_ids_ptr, unique_dst_ids.size(0));
        }

        // Sort the unique_dst_ids tensor.
        auto sorted_unique_dst_ids =
            Sort<false>(unique_dst_ids_ptr, unique_dst_ids.size(0), num_bits);
        auto sorted_unique_dst_ids_ptr =
            sorted_unique_dst_ids.data_ptr<scalar_t>();

        // Mark dst nodes in the src_ids tensor.
        auto is_dst = allocator.AllocateStorage<bool>(src_ids.size(0));
        THRUST_CALL(
            binary_search, sorted_unique_dst_ids_ptr,
            sorted_unique_dst_ids_ptr + unique_dst_ids.size(0), src_ids_ptr,
            src_ids_ptr + src_ids.size(0), is_dst.get());

        // Filter the non-dst nodes in the src_ids tensor, hence only_src.
        auto only_src =
            torch::empty(src_ids.size(0), sorted_unique_dst_ids.options());
        {
          auto is_src = thrust::make_transform_iterator(
              is_dst.get(), thrust::logical_not<bool>{});
          cuda::CopyScalar<int64_t> only_src_size;
          CUB_CALL(
              DeviceSelect::Flagged, src_ids_ptr, is_src,
              only_src.data_ptr<scalar_t>(), only_src_size.get(),
              src_ids.size(0));
          stream.synchronize();
          only_src = only_src.slice(0, 0, static_cast<int64_t>(only_src_size));
        }

        // The code block above synchronizes, ensuring safe access to max_id_src
        // and max_id_dst.
        if (num_bits == 0) {
          num_bits = cuda::NumberOfBits(
              1 + std::max(
                      static_cast<scalar_t>(max_id_src),
                      static_cast<scalar_t>(max_id_dst)));
        }

        // Sort the only_src tensor so that we can unique it later.
        auto sorted_only_src = Sort<false>(
            only_src.data_ptr<scalar_t>(), only_src.size(0), num_bits);

        auto unique_only_src =
            torch::empty(only_src.size(0), src_ids.options());
        auto unique_only_src_ptr = unique_only_src.data_ptr<scalar_t>();

        {  // Compute the unique operation on the only_src tensor.
          cuda::CopyScalar<int64_t> unique_only_src_size;
          CUB_CALL(
              DeviceSelect::Unique, sorted_only_src.data_ptr<scalar_t>(),
              unique_only_src_ptr, unique_only_src_size.get(),
              only_src.size(0));
          stream.synchronize();
          unique_only_src = unique_only_src.slice(
              0, 0, static_cast<int64_t>(unique_only_src_size));
        }

        auto real_order = torch::cat({unique_dst_ids, unique_only_src});
        // Sort here so that binary search can be used to lookup new_ids.
        torch::Tensor sorted_order, new_ids;
        std::tie(sorted_order, new_ids) = Sort(real_order, num_bits);
        auto sorted_order_ptr = sorted_order.data_ptr<scalar_t>();
        auto new_ids_ptr = new_ids.data_ptr<int64_t>();
        // Holds the found locations of the src and dst ids in the sorted_order.
        // Later is used to lookup the new ids of the src_ids and dst_ids
        // tensors.
        auto new_dst_ids_loc =
            allocator.AllocateStorage<scalar_t>(dst_ids.size(0));
        THRUST_CALL(
            lower_bound, sorted_order_ptr,
            sorted_order_ptr + sorted_order.size(0), dst_ids_ptr,
            dst_ids_ptr + dst_ids.size(0), new_dst_ids_loc.get());

        cuda::CopyScalar<bool> all_exist;
        // Check if unique_dst_ids includes all dst_ids.
        if (dst_ids.size(0) > 0) {
          thrust::counting_iterator<int64_t> iota(0);
          auto equal_it = thrust::make_transform_iterator(
              iota, EqualityFunc<scalar_t>{
                        sorted_order_ptr, new_dst_ids_loc.get(), dst_ids_ptr});
          all_exist = Min(equal_it, dst_ids.size(0));
          all_exist.record();
        }

        auto new_src_ids_loc =
            allocator.AllocateStorage<scalar_t>(src_ids.size(0));
        THRUST_CALL(
            lower_bound, sorted_order_ptr,
            sorted_order_ptr + sorted_order.size(0), src_ids_ptr,
            src_ids_ptr + src_ids.size(0), new_src_ids_loc.get());

        // Finally, lookup the new compact ids of the src and dst tensors via
        // gather operations.
        auto new_src_ids = torch::empty_like(src_ids);
        THRUST_CALL(
            gather, new_src_ids_loc.get(),
            new_src_ids_loc.get() + src_ids.size(0),
            new_ids.data_ptr<int64_t>(), new_src_ids.data_ptr<scalar_t>());
        // Perform check before we gather for the dst indices.
        if (dst_ids.size(0) > 0 && !static_cast<bool>(all_exist)) {
          throw std::out_of_range("Some ids not found.");
        }
        auto new_dst_ids = torch::empty_like(dst_ids);
        THRUST_CALL(
            gather, new_dst_ids_loc.get(),
            new_dst_ids_loc.get() + dst_ids.size(0),
            new_ids.data_ptr<int64_t>(), new_dst_ids.data_ptr<scalar_t>());
        return std::make_tuple(real_order, new_src_ids, new_dst_ids);
      }));
}

}  // namespace ops
}  // namespace graphbolt
