#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <hiprand/hiprand_kernel.h>
#include <graphbolt/continuous_seed.h>
#include <graphbolt/cuda_ops.h>
#include <graphbolt/cuda_sampling_ops.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>

#include <algorithm>
#include <array>
#include <hipcub/hipcub.hpp>
#if __CUDA_ARCH__ >= 700
#include <cuda/atomic>
#endif  // __CUDA_ARCH__ >= 700
#include <limits>
#include <numeric>
#include <type_traits>

#include "../random.h"
#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

constexpr int BLOCK_SIZE = 128;

inline __device__ int64_t AtomicMax(int64_t* const address, const int64_t val) {
  // To match the type of "::atomicCAS", ignore lint warning.
  using Type = unsigned long long int;  // NOLINT

  static_assert(sizeof(Type) == sizeof(*address), "Type width must match");

  return atomicMax(reinterpret_cast<Type*>(address), static_cast<Type>(val));
}

inline __device__ int32_t AtomicMax(int32_t* const address, const int32_t val) {
  // To match the type of "::atomicCAS", ignore lint warning.
  using Type = int;  // NOLINT

  static_assert(sizeof(Type) == sizeof(*address), "Type width must match");

  return atomicMax(reinterpret_cast<Type*>(address), static_cast<Type>(val));
}

/**
 * @brief Performs neighbor sampling and fills the edge_ids array with
 * original edge ids if sliced_indptr is valid. If not, then it fills the edge
 * ids array with numbers upto the node degree.
 */
template <typename indptr_t, typename indices_t>
__global__ void _ComputeRandomsNS(
    const int64_t num_edges, const indptr_t* const sliced_indptr,
    const indptr_t* const sub_indptr, const indptr_t* const output_indptr,
    const indices_t* const csr_rows, const uint64_t random_seed,
    indptr_t* edge_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(random_seed, i, 0, &rng);

  while (i < num_edges) {
    const auto row_position = csr_rows[i];
    const auto row_offset = i - sub_indptr[row_position];
    const auto output_offset = output_indptr[row_position];
    const auto fanout = output_indptr[row_position + 1] - output_offset;
    const auto rnd =
        row_offset < fanout ? row_offset : hiprand(&rng) % (row_offset + 1);
    if (rnd < fanout) {
      const indptr_t edge_id =
          row_offset + (sliced_indptr ? sliced_indptr[row_position] : 0);
#if __CUDA_ARCH__ >= 700
      ::cuda::atomic_ref<indptr_t, ::cuda::thread_scope_device> a(
          edge_ids[output_offset + rnd]);
      a.fetch_max(edge_id, ::cuda::std::memory_order_relaxed);
#else
      AtomicMax(edge_ids + output_offset + rnd, edge_id);
#endif  // __CUDA_ARCH__
    }

    i += stride;
  }
}

/**
 * @brief Fills the random_arr with random numbers and the edge_ids array with
 * original edge ids. When random_arr is sorted along with edge_ids, the first
 * fanout elements of each row gives us the sampled edges.
 */
template <
    typename float_t, typename indptr_t, typename indices_t, typename weights_t,
    typename edge_id_t>
__global__ void _ComputeRandoms(
    const int64_t num_edges, const indptr_t* const sliced_indptr,
    const indptr_t* const sub_indptr, const indices_t* const csr_rows,
    const weights_t* const sliced_weights, const indices_t* const indices,
    const continuous_seed random_seed, float_t* random_arr,
    edge_id_t* edge_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;
  const auto labor = indices != nullptr;

  while (i < num_edges) {
    const auto row_position = csr_rows[i];
    const auto row_offset = i - sub_indptr[row_position];
    const auto in_idx = sliced_indptr[row_position] + row_offset;
    const auto rnd = random_seed.uniform(labor ? indices[in_idx] : i);
    const auto prob =
        sliced_weights ? sliced_weights[i] : static_cast<weights_t>(1);
    const auto exp_rnd = -__logf(rnd);
    const float_t adjusted_rnd = prob > 0
                                     ? static_cast<float_t>(exp_rnd / prob)
                                     : std::numeric_limits<float_t>::infinity();
    random_arr[i] = adjusted_rnd;
    edge_ids[i] = row_offset;

    i += stride;
  }
}

struct IsPositive {
  template <typename probs_t>
  __host__ __device__ auto operator()(probs_t x) {
    return x > 0;
  }
};

template <typename indptr_t>
struct MinInDegreeFanout {
  const indptr_t* in_degree;
  const int64_t* fanouts;
  size_t num_fanouts;
  __host__ __device__ auto operator()(int64_t i) {
    return static_cast<indptr_t>(
        min(static_cast<int64_t>(in_degree[i]), fanouts[i % num_fanouts]));
  }
};

template <typename indptr_t, typename indices_t>
struct IteratorFunc {
  indptr_t* indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) { return indices + indptr[i]; }
};

template <typename indptr_t>
struct AddOffset {
  indptr_t offset;
  template <typename edge_id_t>
  __host__ __device__ indptr_t operator()(edge_id_t x) {
    return x + offset;
  }
};

template <typename indptr_t, typename indices_t>
struct IteratorFuncAddOffset {
  indptr_t* indptr;
  indptr_t* sliced_indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) {
    return thrust::transform_output_iterator{
        indices + indptr[i], AddOffset<indptr_t>{sliced_indptr[i]}};
  }
};

template <typename indptr_t, typename in_degree_iterator_t>
struct SegmentEndFunc {
  indptr_t* indptr;
  in_degree_iterator_t in_degree;
  __host__ __device__ auto operator()(int64_t i) {
    return indptr[i] + in_degree[i];
  }
};

c10::intrusive_ptr<sampling::FusedSampledSubgraph> SampleNeighbors(
    torch::Tensor indptr, torch::Tensor indices,
    torch::optional<torch::Tensor> nodes, const std::vector<int64_t>& fanouts,
    bool replace, bool layer, bool return_eids,
    torch::optional<torch::Tensor> type_per_edge,
    torch::optional<torch::Tensor> probs_or_mask,
    torch::optional<torch::Tensor> random_seed_tensor,
    float seed2_contribution) {
  TORCH_CHECK(!replace, "Sampling with replacement is not supported yet!");
  // Assume that indptr, indices, nodes, type_per_edge and probs_or_mask
  // are all resident on the GPU. If not, it is better to first extract them
  // before calling this function.
  auto allocator = cuda::GetAllocator();
  auto num_rows =
      nodes.has_value() ? nodes.value().size(0) : indptr.size(0) - 1;
  auto fanouts_pinned = torch::empty(
      fanouts.size(),
      c10::TensorOptions().dtype(torch::kLong).pinned_memory(true));
  auto fanouts_pinned_ptr = fanouts_pinned.data_ptr<int64_t>();
  for (size_t i = 0; i < fanouts.size(); i++) {
    fanouts_pinned_ptr[i] =
        fanouts[i] >= 0 ? fanouts[i] : std::numeric_limits<int64_t>::max();
  }
  // Finally, copy the adjusted fanout values to the device memory.
  auto fanouts_device = allocator.AllocateStorage<int64_t>(fanouts.size());
  CUDA_CALL(hipMemcpyAsync(
      fanouts_device.get(), fanouts_pinned_ptr,
      sizeof(int64_t) * fanouts.size(), hipMemcpyHostToDevice,
      cuda::GetCurrentStream()));
  auto in_degree_and_sliced_indptr = SliceCSCIndptr(indptr, nodes);
  auto in_degree = std::get<0>(in_degree_and_sliced_indptr);
  auto sliced_indptr = std::get<1>(in_degree_and_sliced_indptr);
  auto max_in_degree = torch::empty(
      1,
      c10::TensorOptions().dtype(in_degree.scalar_type()).pinned_memory(true));
  AT_DISPATCH_INDEX_TYPES(
      indptr.scalar_type(), "SampleNeighborsMaxInDegree", ([&] {
        CUB_CALL(
            DeviceReduce::Max, in_degree.data_ptr<index_t>(),
            max_in_degree.data_ptr<index_t>(), num_rows);
      }));
  // Protect access to max_in_degree with a CUDAEvent
  at::cuda::CUDAEvent max_in_degree_event;
  max_in_degree_event.record();
  torch::optional<int64_t> num_edges;
  torch::Tensor sub_indptr;
  if (!nodes.has_value()) {
    num_edges = indices.size(0);
    sub_indptr = indptr;
  }
  torch::optional<torch::Tensor> sliced_probs_or_mask;
  if (probs_or_mask.has_value()) {
    if (nodes.has_value()) {
      torch::Tensor sliced_probs_or_mask_tensor;
      std::tie(sub_indptr, sliced_probs_or_mask_tensor) = IndexSelectCSCImpl(
          in_degree, sliced_indptr, probs_or_mask.value(), nodes.value(),
          indptr.size(0) - 2, num_edges);
      sliced_probs_or_mask = sliced_probs_or_mask_tensor;
      num_edges = sliced_probs_or_mask_tensor.size(0);
    } else {
      sliced_probs_or_mask = probs_or_mask;
    }
  }
  if (fanouts.size() > 1) {
    torch::Tensor sliced_type_per_edge;
    if (nodes.has_value()) {
      std::tie(sub_indptr, sliced_type_per_edge) = IndexSelectCSCImpl(
          in_degree, sliced_indptr, type_per_edge.value(), nodes.value(),
          indptr.size(0) - 2, num_edges);
    } else {
      sliced_type_per_edge = type_per_edge.value();
    }
    std::tie(sub_indptr, in_degree, sliced_indptr) = SliceCSCIndptrHetero(
        sub_indptr, sliced_type_per_edge, sliced_indptr, fanouts.size());
    num_rows = sliced_indptr.size(0);
    num_edges = sliced_type_per_edge.size(0);
  }
  // If sub_indptr was not computed in the two code blocks above:
  if (nodes.has_value() && !probs_or_mask.has_value() && fanouts.size() <= 1) {
    sub_indptr = ExclusiveCumSum(in_degree);
  }
  auto coo_rows = ExpandIndptrImpl(
      sub_indptr, indices.scalar_type(), torch::nullopt, num_edges);
  num_edges = coo_rows.size(0);
  const continuous_seed random_seed = [&] {
    if (random_seed_tensor.has_value()) {
      return continuous_seed(random_seed_tensor.value(), seed2_contribution);
    } else {
      return continuous_seed{RandomEngine::ThreadLocal()->RandInt(
          static_cast<int64_t>(0), std::numeric_limits<int64_t>::max())};
    }
  }();
  auto output_indptr = torch::empty_like(sub_indptr);
  torch::Tensor picked_eids;
  torch::Tensor output_indices;
  torch::optional<torch::Tensor> output_type_per_edge;

  AT_DISPATCH_INDEX_TYPES(
      indptr.scalar_type(), "SampleNeighborsIndptr", ([&] {
        using indptr_t = index_t;
        if (probs_or_mask.has_value()) {  // Count nonzero probs into in_degree.
          GRAPHBOLT_DISPATCH_ALL_TYPES(
              probs_or_mask.value().scalar_type(),
              "SampleNeighborsPositiveProbs", ([&] {
                using probs_t = scalar_t;
                auto is_nonzero = thrust::make_transform_iterator(
                    sliced_probs_or_mask.value().data_ptr<probs_t>(),
                    IsPositive{});
                CUB_CALL(
                    DeviceSegmentedReduce::Sum, is_nonzero,
                    in_degree.data_ptr<indptr_t>(), num_rows,
                    sub_indptr.data_ptr<indptr_t>(),
                    sub_indptr.data_ptr<indptr_t>() + 1);
              }));
        }
        thrust::counting_iterator<int64_t> iota(0);
        auto sampled_degree = thrust::make_transform_iterator(
            iota, MinInDegreeFanout<indptr_t>{
                      in_degree.data_ptr<indptr_t>(), fanouts_device.get(),
                      fanouts.size()});

        // Compute output_indptr.
        CUB_CALL(
            DeviceScan::ExclusiveSum, sampled_degree,
            output_indptr.data_ptr<indptr_t>(), num_rows + 1);

        auto num_sampled_edges =
            cuda::CopyScalar{output_indptr.data_ptr<indptr_t>() + num_rows};

        // Find the smallest integer type to store the edge id offsets. We synch
        // the CUDAEvent so that the access is safe.
        auto compute_num_bits = [&] {
          max_in_degree_event.synchronize();
          return cuda::NumberOfBits(max_in_degree.data_ptr<indptr_t>()[0]);
        };
        if (layer || probs_or_mask.has_value()) {
          const int num_bits = compute_num_bits();
          std::array<int, 4> type_bits = {8, 16, 32, 64};
          const auto type_index =
              std::lower_bound(type_bits.begin(), type_bits.end(), num_bits) -
              type_bits.begin();
          std::array<torch::ScalarType, 5> types = {
              torch::kByte, torch::kInt16, torch::kInt32, torch::kLong,
              torch::kLong};
          auto edge_id_dtype = types[type_index];
          AT_DISPATCH_INTEGRAL_TYPES(
              edge_id_dtype, "SampleNeighborsEdgeIDs", ([&] {
                using edge_id_t = std::make_unsigned_t<scalar_t>;
                TORCH_CHECK(
                    num_bits <= sizeof(edge_id_t) * 8,
                    "Selected edge_id_t must be capable of storing edge_ids.");
                // Using bfloat16 for random numbers works just as reliably as
                // float32 and provides around 30% speedup.
                using rnd_t = hip_bfloat16;
                auto randoms =
                    allocator.AllocateStorage<rnd_t>(num_edges.value());
                auto randoms_sorted =
                    allocator.AllocateStorage<rnd_t>(num_edges.value());
                auto edge_id_segments =
                    allocator.AllocateStorage<edge_id_t>(num_edges.value());
                auto sorted_edge_id_segments =
                    allocator.AllocateStorage<edge_id_t>(num_edges.value());
                AT_DISPATCH_INDEX_TYPES(
                    indices.scalar_type(), "SampleNeighborsIndices", ([&] {
                      using indices_t = index_t;
                      auto probs_or_mask_scalar_type = torch::kFloat32;
                      if (probs_or_mask.has_value()) {
                        probs_or_mask_scalar_type =
                            probs_or_mask.value().scalar_type();
                      }
                      GRAPHBOLT_DISPATCH_ALL_TYPES(
                          probs_or_mask_scalar_type, "SampleNeighborsProbs",
                          ([&] {
                            using probs_t = scalar_t;
                            probs_t* sliced_probs_ptr = nullptr;
                            if (sliced_probs_or_mask.has_value()) {
                              sliced_probs_ptr = sliced_probs_or_mask.value()
                                                     .data_ptr<probs_t>();
                            }
                            const indices_t* indices_ptr =
                                layer ? indices.data_ptr<indices_t>() : nullptr;
                            const dim3 block(BLOCK_SIZE);
                            const dim3 grid(
                                (num_edges.value() + BLOCK_SIZE - 1) /
                                BLOCK_SIZE);
                            // Compute row and random number pairs.
                            CUDA_KERNEL_CALL(
                                _ComputeRandoms, grid, block, 0,
                                num_edges.value(),
                                sliced_indptr.data_ptr<indptr_t>(),
                                sub_indptr.data_ptr<indptr_t>(),
                                coo_rows.data_ptr<indices_t>(),
                                sliced_probs_ptr, indices_ptr, random_seed,
                                randoms.get(), edge_id_segments.get());
                          }));
                    }));

                // Sort the random numbers along with edge ids, after
                // sorting the first fanout elements of each row will
                // give us the sampled edges.
                CUB_CALL(
                    DeviceSegmentedSort::SortPairs, randoms.get(),
                    randoms_sorted.get(), edge_id_segments.get(),
                    sorted_edge_id_segments.get(), num_edges.value(), num_rows,
                    sub_indptr.data_ptr<indptr_t>(),
                    sub_indptr.data_ptr<indptr_t>() + 1);

                picked_eids = torch::empty(
                    static_cast<indptr_t>(num_sampled_edges),
                    sub_indptr.options());

                // Need to sort the sampled edges only when fanouts.size() == 1
                // since multiple fanout sampling case is automatically going to
                // be sorted.
                if (type_per_edge && fanouts.size() == 1) {
                  // Ensuring sort result still ends up in
                  // sorted_edge_id_segments
                  std::swap(edge_id_segments, sorted_edge_id_segments);
                  auto sampled_segment_end_it = thrust::make_transform_iterator(
                      iota,
                      SegmentEndFunc<indptr_t, decltype(sampled_degree)>{
                          sub_indptr.data_ptr<indptr_t>(), sampled_degree});
                  CUB_CALL(
                      DeviceSegmentedSort::SortKeys, edge_id_segments.get(),
                      sorted_edge_id_segments.get(), picked_eids.size(0),
                      num_rows, sub_indptr.data_ptr<indptr_t>(),
                      sampled_segment_end_it);
                }

                auto input_buffer_it = thrust::make_transform_iterator(
                    iota, IteratorFunc<indptr_t, edge_id_t>{
                              sub_indptr.data_ptr<indptr_t>(),
                              sorted_edge_id_segments.get()});
                auto output_buffer_it = thrust::make_transform_iterator(
                    iota, IteratorFuncAddOffset<indptr_t, indptr_t>{
                              output_indptr.data_ptr<indptr_t>(),
                              sliced_indptr.data_ptr<indptr_t>(),
                              picked_eids.data_ptr<indptr_t>()});
                constexpr int64_t max_copy_at_once =
                    std::numeric_limits<int32_t>::max();

                // Copy the sampled edge ids into picked_eids tensor.
                for (int64_t i = 0; i < num_rows; i += max_copy_at_once) {
                  CUB_CALL(
                      DeviceCopy::Batched, input_buffer_it + i,
                      output_buffer_it + i, sampled_degree + i,
                      std::min(num_rows - i, max_copy_at_once));
                }
              }));
        } else {  // Non-weighted neighbor sampling.
          picked_eids = torch::zeros(num_edges.value(), sub_indptr.options());
          const auto sort_needed = type_per_edge && fanouts.size() == 1;
          const auto sliced_indptr_ptr =
              sort_needed ? nullptr : sliced_indptr.data_ptr<indptr_t>();

          const dim3 block(BLOCK_SIZE);
          const dim3 grid(
              (std::min(num_edges.value(), static_cast<int64_t>(1 << 20)) +
               BLOCK_SIZE - 1) /
              BLOCK_SIZE);
          AT_DISPATCH_INDEX_TYPES(
              indices.scalar_type(), "SampleNeighborsIndices", ([&] {
                using indices_t = index_t;
                // Compute row and random number pairs.
                CUDA_KERNEL_CALL(
                    _ComputeRandomsNS, grid, block, 0, num_edges.value(),
                    sliced_indptr_ptr, sub_indptr.data_ptr<indptr_t>(),
                    output_indptr.data_ptr<indptr_t>(),
                    coo_rows.data_ptr<indices_t>(), random_seed.get_seed(0),
                    picked_eids.data_ptr<indptr_t>());
              }));

          picked_eids =
              picked_eids.slice(0, 0, static_cast<indptr_t>(num_sampled_edges));

          // Need to sort the sampled edges only when fanouts.size() == 1
          // since multiple fanout sampling case is automatically going to
          // be sorted.
          if (sort_needed) {
            const int num_bits = compute_num_bits();
            std::array<int, 4> type_bits = {8, 15, 31, 63};
            const auto type_index =
                std::lower_bound(type_bits.begin(), type_bits.end(), num_bits) -
                type_bits.begin();
            std::array<torch::ScalarType, 5> types = {
                torch::kByte, torch::kInt16, torch::kInt32, torch::kLong,
                torch::kLong};
            auto edge_id_dtype = types[type_index];
            AT_DISPATCH_INTEGRAL_TYPES(
                edge_id_dtype, "SampleNeighborsEdgeIDs", ([&] {
                  using edge_id_t = scalar_t;
                  TORCH_CHECK(
                      num_bits <= sizeof(edge_id_t) * 8,
                      "Selected edge_id_t must be capable of storing "
                      "edge_ids.");
                  auto picked_offsets = picked_eids.to(edge_id_dtype);
                  auto sorted_offsets = torch::empty_like(picked_offsets);
                  CUB_CALL(
                      DeviceSegmentedSort::SortKeys,
                      picked_offsets.data_ptr<edge_id_t>(),
                      sorted_offsets.data_ptr<edge_id_t>(), picked_eids.size(0),
                      num_rows, output_indptr.data_ptr<indptr_t>(),
                      output_indptr.data_ptr<indptr_t>() + 1);
                  auto edge_id_offsets = ExpandIndptrImpl(
                      output_indptr, picked_eids.scalar_type(), sliced_indptr,
                      picked_eids.size(0));
                  picked_eids = sorted_offsets.to(picked_eids.scalar_type()) +
                                edge_id_offsets;
                }));
          }
        }

        output_indices = torch::empty(
            picked_eids.size(0),
            picked_eids.options().dtype(indices.scalar_type()));

        // Compute: output_indices = indices.gather(0, picked_eids);
        AT_DISPATCH_INDEX_TYPES(
            indices.scalar_type(), "SampleNeighborsOutputIndices", ([&] {
              using indices_t = index_t;
              THRUST_CALL(
                  gather, picked_eids.data_ptr<indptr_t>(),
                  picked_eids.data_ptr<indptr_t>() + picked_eids.size(0),
                  indices.data_ptr<indices_t>(),
                  output_indices.data_ptr<indices_t>());
            }));

        if (type_per_edge) {
          // output_type_per_edge = type_per_edge.gather(0, picked_eids);
          // The commented out torch equivalent above does not work when
          // type_per_edge is on pinned memory. That is why, we have to
          // reimplement it, similar to the indices gather operation above.
          auto types = type_per_edge.value();
          output_type_per_edge = torch::empty(
              picked_eids.size(0),
              picked_eids.options().dtype(types.scalar_type()));
          AT_DISPATCH_INTEGRAL_TYPES(
              types.scalar_type(), "SampleNeighborsOutputTypePerEdge", ([&] {
                THRUST_CALL(
                    gather, picked_eids.data_ptr<indptr_t>(),
                    picked_eids.data_ptr<indptr_t>() + picked_eids.size(0),
                    types.data_ptr<scalar_t>(),
                    output_type_per_edge.value().data_ptr<scalar_t>());
              }));
        }
      }));

  // Convert output_indptr back to homo by discarding intermediate offsets.
  output_indptr =
      output_indptr.slice(0, 0, output_indptr.size(0), fanouts.size());
  torch::optional<torch::Tensor> subgraph_reverse_edge_ids = torch::nullopt;
  if (return_eids) subgraph_reverse_edge_ids = std::move(picked_eids);
  if (!nodes.has_value()) {
    nodes = torch::arange(indptr.size(0) - 1, indices.options());
  }

  return c10::make_intrusive<sampling::FusedSampledSubgraph>(
      output_indptr, output_indices, nodes.value(), torch::nullopt,
      subgraph_reverse_edge_ids, output_type_per_edge);
}

}  //  namespace ops
}  //  namespace graphbolt
